#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>
#include <algorithm>
#include <math.h>
#include <cstdlib>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

using namespace std;

struct cidade {
    int id;
    double x;
    double y;
};

float Dist(cidade a, cidade b){
    float totalDist = 0;
    totalDist = pow(pow(a.x - b.x, 2) + pow(a.y - b.y, 2), 0.5);
    return totalDist;
}

struct saxpy
{
    int *a;
    float *b;
    int c;
    saxpy(int *a_, float *b_, int c_) : a(a_), b(b_), c(c_) {};
    __host__ __device__
    float operator()(const int x) {
        int cont = x*c;
        float dist = 0;
        float melhorDist = 0;
        int temp = 0;
        bool check = true;
        for(int i = 0; i < (c-1); i++){
            melhorDist += b[a[cont+i]*c + a[cont+i+1]];
        }
        melhorDist += b[a[cont+(c-1)]*c + a[cont+0]];
        while(check){
            check = false;
            for(int e = 0; e < (c-1); e++){
                temp = a[cont+e+1];
                a[cont+e+1] = a[cont+e];
                a[cont+e] = temp;
                for(int i = 0; i < (c-1); i++){
                    dist += b[a[cont+i]*c + a[cont+i+1]];
                }
                dist += b[a[cont+(c-1)]*c + a[cont+0]];
                if(dist < melhorDist){
                    melhorDist = dist;
                    check = true;
                } else {
                    temp = a[cont+e+1];
                    a[cont+e+1] = a[cont+e];
                    a[cont+e] = temp; 
                }
                dist = 0;
            }
        }
        return melhorDist;
    }
};

int main(){
    int numCidades;
    cin >> numCidades;
    vector<cidade> cidades;
    for(int i = 0; i < numCidades; i++){
        cidade novo;
        novo.id = i;
        double texto;
        cin >> texto;
        novo.x = texto;
        cin >> texto;
        novo.y = texto;
        cidades.push_back(novo);
    }

    thrust::host_vector<float> Distancias(numCidades*numCidades);

    for(int i = 0; i < numCidades; i++){
        for(int e = 0; e < numCidades; e++){
            Distancias[(cidades[e].id*numCidades) + cidades[i].id] = Dist(cidades[i], cidades[e]);
        }
    }

    thrust::device_vector<float> gpuDist(numCidades*numCidades);

    gpuDist = Distancias;

    thrust::host_vector<int> ordens(numCidades*numCidades*10);
    srand(1);
    int counter = 0;
    int counterGlobal = 0;
    vector<int> randoms;
    for(int i = 0; i < numCidades*10; i++){
        while(counter < numCidades){
            int index = rand() % numCidades;
            bool found = (find(randoms.begin(), randoms.end(), index) != randoms.end());
            if(found != 1){
                randoms.push_back(index);
                ordens[counterGlobal] = index;
                counter += 1;
                counterGlobal += 1;
            }
        }
        randoms.clear();
        counter = 0;
    }

    thrust::device_vector<int> ordensGPU(numCidades*numCidades*10);

    ordensGPU = ordens;
    thrust::device_vector<float> data(numCidades*10);

    auto iter = thrust::make_counting_iterator(0);


    thrust::transform(iter, iter+numCidades*10, data.begin(), saxpy(thrust::raw_pointer_cast(ordensGPU.data()), thrust::raw_pointer_cast(gpuDist.data()), numCidades));

    float resultPrimeiro = data[0];

    float result = thrust::reduce(data.begin(), data.end(), resultPrimeiro, thrust::minimum<float>());

    cout << result << endl;

    return 0;
}